#include "hip/hip_runtime.h"
/*
 * Copyright 2017 of original authors and authors.
 *
 * We use MIT license for this project, checkout LICENSE file in the root of source tree.
 */

#include <stdio.h>
#include <rte_mbuf.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <>
#include <iostream>
#include <rte_ip.h>
#include "cuda_sync_lcore_function.h"

namespace gpuflow {
namespace cu {

__device__ void IPv4Processing(struct ipv4_hdr *ipv4_header, struct ether_hdr *ether_header) {
  printf("Dealing with ipv4 header!\n");
}

__device__ void IPv6Processing(struct ipv6_hdr *ipv6_header, struct ether_hdr *ether_header) {
  printf("Dealing with ipv6 header!\n");
}

__device__ void EtherCopy(struct ether_hdr *ether_header) {

}

__global__ void PacketProcessing(uint8_t *dev_ptypes_burst,
                                 struct ipv4_hdr *dev_ipv4_hdr_burst,
                                 struct ipv6_hdr *dev_ipv6_hdr_burst,
                                 struct ether_hdr *dev_ether_hdrs_burst,
                                 int nb_of_ip_hdrs) {
  int idx = threadIdx.x;
  if (idx < nb_of_ip_hdrs) {
    // Match up packet types.
    if(dev_ptypes_burst[idx] == IP_FAMILY::PTYPE_IPV4) {
      IPv4Processing(&dev_ipv4_hdr_burst[idx], &dev_ether_hdrs_burst[idx]);
    } else if (dev_ptypes_burst[idx] == IP_FAMILY::PTYPE_IPV6) {
      IPv6Processing(&dev_ipv6_hdr_burst[idx], &dev_ether_hdrs_burst[idx]);
    } else {
      // ignore
    }
  }
}

// Setup cuda devices
int CudaSyncLCoreFunction::SetupCudaDevices(int nb_rx) {

  // Allocate pointers.
  hipError_t error;
  error = hipMalloc((void **)&dev_ptypes_burst, nb_rx * sizeof(uint8_t));
  if (error != hipSuccess) {
    std::cerr << "Device memory allocation failed, abort." << std::endl;
    exit(1);
  }
  error = hipMalloc((void **)&dev_ipv4_hdrs_burst, nb_rx * sizeof(struct ipv4_hdr));
  if (error != hipSuccess) {
    std::cerr << "Device memory allocation failed, abort." << std::endl;
    exit(1);
  }
  error = hipMalloc((void **)&dev_ipv6_hdrs_burst, nb_rx * sizeof(struct ipv6_hdr));
  if (error != hipSuccess) {
    std::cerr << "Device memory allocation failed, abort." << std::endl;
    exit(1);
  }
  error = hipMalloc((void **)&dev_ether_hdrs_burst, nb_rx * sizeof(struct ether_hdr));
  if (error != hipSuccess) {
    std::cerr << "Device memory allocation failed, abort." << std::endl;
    exit(1);
  }

  return 0;
}

int CudaSyncLCoreFunction::ProcessPacketsBatch(struct rte_mbuf **pkts_burst, int nb_rx) {
  SetupCudaDevices(nb_rx);
  hipError_t error;
  for (int i = 0; i < nb_rx; ++i) {
    if (RTE_ETH_IS_IPV4_HDR(pkts_burst[i]->packet_type)) {
      // Ipv4 header, copy ipv4
      error = hipMemcpy(&dev_ipv4_hdrs_burst[i], rte_pktmbuf_mtod_offset(pkts_burst[i], struct ipv4_hdr *, sizeof(struct ether_hdr)),
                         sizeof(struct ipv4_hdr), hipMemcpyHostToDevice);
      if (error != hipSuccess) {
        std::cerr << "Memory copy error on cuda mem copy" << std::endl;
        exit(1);
      }
      // Add type into type burst.
      error = hipMemcpy(&dev_ptypes_burst[i], &IP_FAMILY::PTYPE_IPV4, sizeof(uint8_t), hipMemcpyHostToDevice);
      if (error != hipSuccess) {
        std::cerr << "Memory copy error on cuda mem copy" << std::endl;
        exit(1);
      }
      // Copy ether header
      error = hipMemcpy(&dev_ether_hdrs_burst[i], rte_pktmbuf_mtod(pkts_burst[i], struct ether_hdr *),
                         sizeof(struct ether_hdr), hipMemcpyHostToDevice);
      if (error != hipSuccess) {
        std::cerr << "Memory copy error on cuda mem copy" << std::endl;
        exit(1);
      }
    } else if (RTE_ETH_IS_IPV6_HDR(pkts_burst[i]->packet_type)) {
      // Ipv6 header, copy ipv6 type
      error = hipMemcpy(&dev_ipv6_hdrs_burst[i], rte_pktmbuf_mtod_offset(pkts_burst[i], struct ipv6_hdr *, sizeof(struct ether_hdr)),
                         sizeof(struct ipv6_hdr), hipMemcpyHostToDevice);
      if (error != hipSuccess) {
        std::cerr << "Memory copy error on cuda mem copy" << std::endl;
        exit(1);
      }
      // Add type into type burst.
      error = hipMemcpy(&dev_ptypes_burst[i], &IP_FAMILY::PTYPE_IPV6, sizeof(uint8_t), hipMemcpyHostToDevice);
      if (error != hipSuccess) {
        std::cerr << "Memory copy error on cuda mem copy" << std::endl;
        exit(1);
      }
      // Copy ether header
      error = hipMemcpy(&dev_ether_hdrs_burst[i], rte_pktmbuf_mtod(pkts_burst[i], struct ether_hdr *),
                         sizeof(struct ether_hdr), hipMemcpyHostToDevice);
      if (error != hipSuccess) {
        std::cerr << "Memory copy error on cuda mem copy" << std::endl;
        exit(1);
      }
    } else {
      struct ether_hdr *ether_header = rte_pktmbuf_mtod(pkts_burst[i], struct ether_hdr *);
      if (ether_header->ether_type == rte_cpu_to_be_16(ETHER_TYPE_ARP)) {
        // is arp packet.
        std::cout << "ARP!" << std::endl;
      }
      // Continue, the index will be jumped off on either ethernet header burst, ipv4 header burst or ipv6 header burst.
    }
  }

  PacketProcessing<<<1, nb_rx>>>(dev_ptypes_burst,
          dev_ipv4_hdrs_burst,
          dev_ipv6_hdrs_burst,
          dev_ether_hdrs_burst,
          nb_rx);

  hipFree(dev_ptypes_burst);
  hipFree(dev_ether_hdrs_burst);
  hipFree(dev_ipv4_hdrs_burst);
  hipFree(dev_ipv6_hdrs_burst);
  return 0;
}

} // namespace cu
} // namespace gpuflow

